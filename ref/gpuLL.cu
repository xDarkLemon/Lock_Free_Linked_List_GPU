#include <stdio.h>
#include <hip/hip_runtime.h>

// compile with nvcc -o ll gpuLL.cu

// the first 3 bits of a pointer are empty, use the first bit as marker
#define IS_MARKED(p)  ((int)(((unsigned long long)(p)) & 1))
#define GET_MARKED_REF(p) (((unsigned long long)(p)) | 1)
#define GET_UNMARKED_REF(p) (((unsigned long long)(p)) & ~1)

struct node {
	int data;
	struct node *next;
};

__device__ struct node *head;

__device__ struct node *createNode(int val) {
	struct node *newnode = (struct node *)malloc(sizeof(struct node));
	newnode->data = val;
	newnode->next = NULL;
	return newnode;
}

__global__ void listInit()
{
	head = createNode(-1);
	struct node *tail = createNode(-1);
	head->next=tail;
}

__device__ void addFront(struct node *newnode)
{
	newnode->next = head->next;
	head->next = newnode;
}

__global__ void addFront(int val)
{
	// need to modify
	struct node *newnode=createNode(val);
	addFront(newnode);
}

__device__ void nodePrint(struct node *ptr) {
	if (ptr->data==-1)
		if(ptr->next)
			printf("head ");
		else
			printf("tail\n");
	else
		printf("%d ", ptr->data);
}

__global__ void listPrint() {
	printf("listPrint\n");
	int nnodes = 0;
	for (struct node *ptr = head; ptr; ptr = (struct node *)GET_UNMARKED_REF(ptr->next), ++nnodes)
	{
		// printf("ptr: %llu, ",GET_UNMARKED_REF(ptr));
		nodePrint(ptr);
	}
	printf("Number of nodes = %d\n", nnodes);
}

__global__ void listPrintRaw() {
	// print with marked nodes
	printf("listPrintRaw\n");
	int nnodes = 0;
	for (struct node *ptr = head; ptr; ptr = (struct node *)GET_UNMARKED_REF(ptr->next))
	{	
		// printf("ptr: %llu, ",GET_UNMARKED_REF(ptr));
		if(!IS_MARKED(ptr->next))
		{
			nodePrint(ptr);
			nnodes++;
		}
	}
	printf("Number of nodes = %d\n", nnodes);
}

__device__ void printVal(int val)
{
	printf("val: %d\n",val);
}

__global__ void printVal(int *arr, int N)
{
	for (int i=0;i<N;i++)
	{
		printVal(arr[i]);
	}
}

__device__ struct node *searchNode(int val)
{
	struct node *cur;
	for (struct node *ptr = head; ptr->next; ptr = (struct node *)GET_UNMARKED_REF(ptr->next))
	{
		cur = (struct node *)GET_UNMARKED_REF(ptr->next);
		if (cur->data == val)
		{
			return cur;
		}
	}
	return NULL;
}

__device__ struct node *listSearch(int val)
{
	printf("listSearch val: %d\n", val);
	struct node *cur=NULL, *p, *prev_next;
	struct node *prev;
	int cnt = 0;
	while(1)
	{
		// step1: traverse the list and find the node
		for(cur=head; cur->next; cur=(struct node *)GET_UNMARKED_REF(cur->next))
		{
			if(IS_MARKED(cur->next))  // p->next is marked means p is deleted logically
			{
				// printf("next is marked\n");
				continue;  // skip this node
			}
			if(cur->data == val)  // found
			{
				// cur = p;
				// printf("cur data %d found\n", cur->data);
				break;
			}
			prev=cur;

		}
		if(cur->next==NULL)  // cur is the tail node
		{
			printf("%d not found\n", val);
			printf("prev data: %d, prev->next data: %d\n", prev->data, prev->next->data);
			// break;  // now break, future point cur to tail node
		}
		else
			printf("val found, cur->data: %d, cur ref: %llu\n", cur->data, GET_UNMARKED_REF(cur));
		// breaks;

		// no marked nodes between prev and cur
		if (prev->next == cur)
		{
			if (!cur->next)  // cur not found, cur is tail node
			{
				printf("cur reaches the tail\n");
				break;  // then return cur
			}
			else
				if (!IS_MARKED(cur->next))  // if cur is marked as removed during the time, search again
					break;  // then return cur
		}
		
		// step2: remove marked nodes between prev and cur
		else
		{
			// printf("prev data: %d, prev->next data: %d, cur data: %d\n", prev->data, (prev->next)->data, cur->data);
            
			// Step 2.1: If an insertions was made in the meantime between left and right, repeat search.
			int inserted = 0;
			for(p=(struct node *)GET_UNMARKED_REF(prev->next); p==cur; p=(struct node *)GET_UNMARKED_REF(p->next))
			{
				// loop from prev to cur, if there is any unmarked node, it is inserted meantime, need to search again
                if (!IS_MARKED(p->next))
					inserted = 1;
			}
			if (inserted==1)
				continue;  // search again
			
			// No unmarked nodes in between now
			// Step 2.2: Try to "remove" the marked nodes between left and right.
			prev_next = (struct node *)atomicCAS((unsigned long long *)&prev->next, GET_UNMARKED_REF(prev->next), (unsigned long long)cur);  
			// update prev->next to cur, delete marked nodes in between (no garbage collection yet)
            if(prev_next!=(struct node *)GET_UNMARKED_REF(prev->next))
			{
				if(!prev_next) printf("prev_next NULL\n");
				else printf("prev_next->data: %d\n",prev_next->data);
				if(!prev->next) printf("prev->next NULL\n");
				// somone changed left->next, deletion failed, search again
				continue;
			}
        }
	}
	return cur;
}

__global__ void listSearchOne(int val)
{
	listSearch(val);
	// printf("\nFind node %d\nunmarked addr: %llu, marked addr: %llu, data: %d\n", val, GET_UNMARKED_REF(p), GET_MARKED_REF(p), ((struct node *)GET_UNMARKED_REF(p))->data);
}

__device__ void listTraverseDel()
{
	struct node *cur, *prev, *p, *prev_next;
	prev=head;
	cur=head->next;
	for(cur=head->next; cur->next; cur=(struct node*)GET_UNMARKED_REF(cur->next))
	{
		if(IS_MARKED(cur->next))  // p->next is marked means p is deleted logically
		{
			continue;  // skip this node
		}
		if(prev->next!=cur)  // stop here and do deletion
		{
			printf("prev: %d, cur: %d\n", prev->data, cur->data);
			prev->next=cur;
		}
		prev=cur;
	}
}

__global__ void listTraverse()
{
	// delete marked nodes during traversal
	listTraverseDel();
}

__global__ void listInsert(int *insertVals, int *insertPrevs, int N) {
	// insert ater a certain value
	unsigned idx = blockIdx.x * blockDim.x + threadIdx.x; 
	if (idx<N)
	{
		struct node *myold, *actualold;
		struct node *prev = listSearch(insertPrevs[idx]);
		// struct node *prev = searchNode(insertPrevs[idx]);
		if (prev)
		{
			struct node *newnode = createNode(insertVals[idx]);

			do {
				myold = prev->next;  // should reload every iteration
				newnode->next = myold;
				actualold = (struct node *)atomicCAS((unsigned long long *)&prev->next, (unsigned long long)myold, (unsigned long long)newnode);  
			} while (actualold != myold);
		}
		else
			printf("Prev %d not found\n", insertPrevs[idx]);
	}
}

__global__ void listRemove(int *Vals, int N)
{
	unsigned idx = blockIdx.x * blockDim.x + threadIdx.x; 
	if (idx<N)
	{
		int val = Vals[idx];
		// printf("thread idx: %d, remove val: %d\n", idx, val);
		struct node *prev, *cur, *succ, *actual_succ;
		prev = cur = succ = NULL;
		int cnt=0;

		while(1)
		{
			// printf("cnt: %d\n", cnt++);
			cur = listSearch(val);
			// cur = listSearch(val, &prev);  // question: why prev is not used later?
			// cur = searchNode(val);
			// cur = searchNode(val, &prev);
			// printf("cur ptr: %llu\n", (unsigned long long) cur);
			if (cur==NULL || cur->data != val)
			{
				// printf("Remove node %d not found\n", val);
				break;
			}
			else
			{
				succ = cur->next;
				if(!IS_MARKED(succ))
				{
					actual_succ = (struct node *)atomicCAS((unsigned long long *)&cur->next, (unsigned long long)succ, GET_MARKED_REF(succ));  // actual cur->next set as marked succ
					if(actual_succ==succ)
					{
/*
						printf("Remove found %d\n", val);
						printf("unmarked succ: %llu, marked succ: %llu, succ: %llu, actual succ: %llu, cur->next: %llu\n", GET_UNMARKED_REF(succ), GET_MARKED_REF(succ), (unsigned long long)succ, (unsigned long long)actual_succ, (unsigned long long)cur->next);
						for (struct node *ptr = head; ptr; ptr = (struct node *)GET_UNMARKED_REF(ptr->next))
						{
							if (!ptr->data)
								printf("head ");
							else
							{
								if(!IS_MARKED(ptr->next))
									printf("%d ", ptr->data);
							}
						}
						printf("\n");
*/
						break;
					}
				}
			}
		}
		// listPrintRawDev();
	}
}

void Demo() {
	printf("listInit\n");
	listInit<<<1,1>>>();
	addFront<<<1,1>>>(3);
	addFront<<<1,1>>>(2);
	addFront<<<1,1>>>(1);
	listPrint<<<1, 1>>>();
	hipDeviceSynchronize();

	int *insert_h = (int *)malloc(sizeof(int)*5);
	insert_h[0]=50;
	insert_h[1]=60;
	insert_h[2]=70;
	insert_h[3]=80;
	insert_h[4]=90;
	int *insert_d;
	hipMalloc((void **)&insert_d, sizeof(int)*5);
	hipMemcpy(insert_d, insert_h, sizeof(int)*5, hipMemcpyHostToDevice);
	// printf("Insert vals\n");
	// printVal<<<1,1>>>(insert_d, 5);
	// cudaDeviceSynchronize();

	int *prev_h = (int *)malloc(sizeof(int)*5);
	prev_h[0]=2;
	prev_h[1]=2;
	prev_h[2]=2;
	prev_h[3]=1;
	prev_h[4]=3;
	int *prev_d;
	hipMalloc((void **)&prev_d, sizeof(int)*5); 
	hipMemcpy(prev_d, prev_h, sizeof(int)*5, hipMemcpyHostToDevice);
	// printf("Insert prevs\n");
	// printVal<<<1,1>>>(prev_d, 5);
	// cudaDeviceSynchronize();

	int *rm_h = (int *)malloc(sizeof(int)*3);
	rm_h[0]=1;
	rm_h[1]=80;
	rm_h[2]=70;
	int *rm_d;
	hipMalloc((void **)&rm_d, sizeof(int)*3); 
	hipMemcpy(rm_d, rm_h, sizeof(int)*3, hipMemcpyHostToDevice);
	// printf("Remove vals\n");
	// printVal<<<1,1>>>(rm_d, 3);
	// cudaDeviceSynchronize();
	
	printf("\nlistInsert\n");
	listInsert<<<4, 4>>>(insert_d, prev_d, 5);
	hipDeviceSynchronize();
	listPrint<<<1, 1>>>();
	hipDeviceSynchronize();
	
	printf("\nlistRemove\n");
	listRemove<<<1, 4>>>(rm_d, 3);
	hipDeviceSynchronize();  // necessary!
	listPrintRaw<<<1, 1>>>();
	hipDeviceSynchronize();  // necessary!	
	listPrint<<<1, 1>>>();
	hipDeviceSynchronize();  // necessary!
	
	printf("\nlistTraverse\n");
	// listSearchOne<<<1,1>>>(80);
	listTraverse<<<1,1>>>();
	hipDeviceSynchronize();  // necessary!
	listPrintRaw<<<1, 1>>>();
	hipDeviceSynchronize();  // necessary!	
	listPrint<<<1, 1>>>();
	hipDeviceSynchronize();  // necessary!
}

void parallelOperate(const int *Nodes, const int N, const int *ops, const int *opNodes, const int *insertNodes, const int opN) 
{
}

int main()
{
	Demo();
	return 0;
}